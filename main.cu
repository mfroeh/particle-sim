#include "hip/hip_runtime.h"
#define GLFW_INCLUDE_NONE
#include <GLFW/glfw3.h>
#include "glad/glad.h"
#include <iostream>
#include <array>
#include <vector>
#include <random>
#include <cassert>

using namespace std;

enum class Color
{
    RED = 0,
    GREEN = 1,
    BLUE = 2,
    YELLOW = 3,
    MAGENTA = 4,
    CYAN = 5,
    MAX,
};

struct Pixel
{
    unsigned char r, g, b, a{255};

    Pixel() : r{0}, g{0}, b{0} {}
    Pixel(unsigned char r, unsigned char g, unsigned char b) : r{r}, g{g}, b{b} {}
    Pixel(unsigned char r, unsigned char g, unsigned char b, unsigned char a) : r{r}, g{g}, b{b}, a{a} {}
    Pixel(Color color)
    {
        switch (color)
        {
        case Color::RED:
            r = 255;
            g = 0;
            b = 0;
            break;
        case Color::GREEN:
            r = 0;
            g = 255;
            b = 0;
            break;
        case Color::BLUE:
            r = 0;
            g = 0;
            b = 255;
            break;
        case Color::YELLOW:
            r = 255;
            g = 255;
            b = 0;
            break;
        case Color::MAGENTA:
            r = 255;
            g = 0;
            b = 255;
            break;
        case Color::CYAN:
            r = 0;
            g = 255;
            b = 255;
            break;
        }
    }

    Pixel operator*(double val)
    {
        return Pixel(r * val, g * val, b * val, a * val);
    }
};

struct Particle
{
    double x{}, y{};
    double velX{}, velY{};
    Color color{};
};

__device__ double getForce(double distanceRatio, double attraction)
{
    const double repulsiveRadius{0.25};
    if (distanceRatio < repulsiveRadius)
    {
        return distanceRatio / repulsiveRadius - 1;
    }
    else
    {
        return attraction * (1 - distanceRatio);
    }
}

double const RADIUS{0.2};
size_t const colorCount{static_cast<size_t>(Color::MAX)};

__global__ void updatePosition(Particle *particles, size_t particleCount, double *attractionMatrix)
{
    double const FRICTION{pow(.5, 10)};

    unsigned idx{blockIdx.x * blockDim.x + threadIdx.x};

    Particle &me{particles[idx]};

    double forceX{}, forceY{};
    for (size_t i{}; i < particleCount; ++i)
    {
        if (i == idx)
            continue;
        Particle const &other{particles[i]};
        double x{other.x - me.x};
        double y{other.y - me.y};
        double dist{sqrt(x * x + y * y)};
        dist = max(dist, 0.0000000001); // Avoid division by zero

        if (dist < RADIUS)
        {
            double attraction{attractionMatrix[static_cast<int>(me.color) * colorCount + static_cast<int>(other.color)]};
            double force{getForce(dist / RADIUS, attraction)};

            forceX += (x / dist) * force;
            forceY += (y / dist) * force;
        }
    }

    me.velX += forceX;
    me.velY += forceY;
    me.velX *= FRICTION;
    me.velY *= FRICTION;

    me.x += me.velX;
    me.y += me.velY;
}

void draw(vector<Particle> const &particles, Pixel *image, int width, int height)
{
    for (size_t i{}; i < height; ++i)
    {
        for (size_t j{}; j < width; ++j)
        {
            image[i * width + j] = {0, 0, 0, 0};
        }
    }

    for (Particle const &particle : particles)
    {
        int imageX{particle.x * width};
        int imageY{particle.y * height};
        if (imageX > 0 && imageX < width && imageY > 0 && imageY < height)
            image[imageY * width + imageX] = Pixel{particle.color};
    }
}

int main(int argc, char *argv[])
{
    if (!glfwInit())
        return 1;

    GLFWwindow *window = glfwCreateWindow(1024, 1024, "Particle simulation", NULL, NULL);
    if (!window)
        return 1;

    glfwMakeContextCurrent(window);
    if (!gladLoadGLLoader(reinterpret_cast<GLADloadproc>(glfwGetProcAddress)))
        return 1;

    glfwSetWindowSizeCallback(window, [](GLFWwindow *win, int width, int height)
                              { glViewport(0, 0, width, height); });
    glViewport(0, 0, 1024, 1024);

    int seed{12};
    if (argc >= 2)
        seed = stoi(argv[1]);
    default_random_engine e{seed};

    // attraction matrix from x to
    uniform_real_distribution<double> uniform{-1, 1};
    size_t const colorCount{static_cast<size_t>(Color::MAX)};
    array<double, colorCount * colorCount> attractionMatrix{-0.5, 0.5, -0.5, -0.5, 0.5, -0.5, 0.5, 0.5, 0.5};
    for (size_t i{}; i < colorCount * colorCount; ++i)
        attractionMatrix[i] = uniform(e);
    double *devAttractionMatrix;
    hipMalloc(&devAttractionMatrix, sizeof(double) * colorCount * colorCount);
    hipMemcpy(devAttractionMatrix, attractionMatrix.data(), sizeof(double) * colorCount * colorCount, hipMemcpyHostToDevice);

    // particles
    uniform = uniform_real_distribution<double>{0, 1};
    size_t particleCount{4 * 1024};
    if (argc >= 3)
        particleCount = stoi(argv[2]);
    assert(particleCount % 1024 == 0);
    vector<Particle> particles{particleCount};
    for (size_t i{}; i < particleCount; ++i)
    {
        particles[i].x = uniform(e);
        particles[i].y = uniform(e);
        particles[i].color = static_cast<Color>(i % colorCount);
    }
    Particle *devParticles;
    hipMalloc(&devParticles, sizeof(Particle) * particleCount);

    cout << "Simulation: " << particleCount << " particles, " << colorCount << " colors, " << seed << " is the seed" << endl;
    cout << "Sampled attraction matrix:" << endl;
    for (size_t i{}; i < colorCount; ++i)
    {
        for (size_t j{}; j < colorCount; ++j)
        {
            cout << attractionMatrix[i * colorCount + j] << " ";
        }
        cout << endl;
    }

    int width{0}, height{0};
    Pixel *image = new Pixel[1];
    unsigned frames = 0;
    double prevTime = glfwGetTime();
    while (!glfwWindowShouldClose(window))
    {
        glfwPollEvents();
        int prevWidth{width}, prevHeight{height};
        glfwGetFramebufferSize(window, &width, &height);
        if (width != prevWidth || height != prevHeight)
        {
            delete image;
            image = new Pixel[width * height];
        }

        hipMemcpy(devParticles, particles.data(), sizeof(Particle) * particleCount, hipMemcpyHostToDevice);

        dim3 blockDim{1024};
        dim3 gridDim{particleCount / 1024};
        updatePosition<<<gridDim, blockDim>>>(devParticles, particleCount, devAttractionMatrix);
        hipDeviceSynchronize();
        hipMemcpy(particles.data(), devParticles, sizeof(Particle) * particleCount, hipMemcpyDeviceToHost);

        draw(particles, image, width, height);

        // actually draw
        glClearColor(0.f, 0.f, 0.f, 1.f);
        glClear(GL_COLOR_BUFFER_BIT);

        glDrawPixels(width, height, GL_RGBA, GL_UNSIGNED_BYTE, image);
        glfwSwapBuffers(window);

        // fps
        frames++;
        double time{glfwGetTime()};
        double delta{time - prevTime};
        if (delta >= 1.)
        {
            double fps = frames / delta;
            glfwSetWindowTitle(window, ("Particle simulation, FPS: " + to_string(fps)).c_str());
            prevTime = time;
            frames = 0;
        }
    }

    // TODO: Use RAII
    glfwDestroyWindow(window);
    glfwTerminate();
}
